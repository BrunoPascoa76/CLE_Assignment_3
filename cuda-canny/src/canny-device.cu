#include "hip/hip_runtime.h"
// CLE 24'25
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// utilities for safe cuda api calls copied from cuda sdk.

#define cudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define cudaCheckMsg(msg) __cudaGetLastError(msg, __FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

inline void __cudaGetLastError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : cutilCheckMsg() CUTIL CUDA error : %s : (%d) %s.\n",
                file, line, errorMessage, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

#define max(a, b) (((a) > (b)) ? (a) : (b))
#define min(a, b) (((a) < (b)) ? (a) : (b))

#define MAX_BRIGHTNESS 255

// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

// include image functions
#include "image.c"

// CUDA kernel to generate Gaussian kernel on GPU
__global__ void generate_gaussian_kernel(float *kernel, int n, float sigma)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n * n) return;
    
    int i = idx / n;
    int j = idx % n;
    float mean = (float)floor(n / 2.0);
    
    kernel[idx] = expf(-0.5f * (powf((i - mean) / sigma, 2.0f) + 
                               powf((j - mean) / sigma, 2.0f))) / 
                  (2.0f * M_PI * sigma * sigma);
}

__global__ void convolution_cuda_kernel(const pixel_t *in, pixel_t *out, const float *kernel, const int nx, const int ny, const int kn)
{
    // get current coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int khalf = kn / 2;

    if(x < khalf || y < khalf || x >= nx - khalf || y >= ny - khalf)
        return; // the borders weren't touched in the cpu version, so trying to replicate that

    float sum = 0.0f;

    // now do the actual convolution for this pixel
    for (int ky = -khalf; ky <= khalf; ky++)
    {
        for (int kx = -khalf; kx <= khalf; kx++)
        {
            int ix = x + kx;
            int iy = y + ky;

            if (ix >= 0 && ix < nx && iy >= 0 && iy < ny)
            { // if the current kernel point is whithin bounds...
                float val = in[iy * nx + ix];
                float weight = kernel[(ky + khalf) * kn + (kx + khalf)];
                sum += val * weight;
            }
        }
    }

    out[y * nx + x] = (pixel_t)sum;
}

__global__ void min_max_cuda(const pixel_t *in, const int nx, const int ny, pixel_t *min_val, pixel_t *max_val)
{
    extern __shared__ pixel_t sdata[];

    pixel_t *smin = sdata; //get the pointers for the min and max positions
    pixel_t *smax = &sdata[blockDim.x*blockDim.y];

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize shared memory with extreme values for threads outside image bounds
    if(x >= nx || y >= ny) {
        smin[tid] = INT_MAX;
        smax[tid] = INT_MIN;
    } else {
        pixel_t val = in[y * nx + x];
        smin[tid] = val;
        smax[tid] = val;
    }
    __syncthreads();
    
    // Parallel reduction
    for(int s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1) {
        if(tid < s) {
            smin[tid] = min(smin[tid], smin[tid + s]);
            smax[tid] = max(smax[tid], smax[tid + s]);
        }
        __syncthreads();
    }

    if(tid == 0) {
        atomicMin(min_val, smin[0]);
        atomicMax(max_val, smax[0]);
    }
}

__global__ void normalize_cuda(pixel_t *inout, const int nx, const int ny, const int kn, const int min_val, const int max_val)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int khalf = kn / 2;

    if(x < khalf || y < khalf || x >= nx - khalf || y >= ny - khalf)
        return;

    if (max_val != min_val) {
        pixel_t pixel = (pixel_t)(MAX_BRIGHTNESS * ((float)(inout[y * nx + x] - min_val) / (float)(max_val - min_val)));
        inout[y * nx + x] = pixel;
    } else {
        inout[y * nx + x] = 0;
    }
}

void gaussian_filter_cuda(const pixel_t *in, pixel_t *out, const int nx, const int ny, const float sigma)
{
    const int n = 2 * (int)(2 * sigma) + 3;
    
    fprintf(stderr, "gaussian_filter: kernel size %d, sigma=%g\n", n, sigma);

    // Allocate memory for kernel on GPU
    float *d_kernel;
    cudaSafeCall(hipMalloc((void **)&d_kernel, n * n * sizeof(float)));

    // Generate Gaussian kernel on GPU
    int kernel_threads = 256;
    int kernel_blocks = (n * n + kernel_threads - 1) / kernel_threads;
    generate_gaussian_kernel<<<kernel_blocks, kernel_threads>>>(d_kernel, n, sigma);
    cudaCheckMsg("generate_gaussian_kernel launch failed");

    // Set up grid and block dimensions for convolution
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // Apply Gaussian filter using convolution
    convolution_cuda_kernel<<<grid, block>>>(in, out, d_kernel, nx, ny, n);
    cudaCheckMsg("gaussian convolution launch failed");

    // Find min and max values for normalization
    pixel_t *d_max, *d_min;
    pixel_t h_max = INT_MIN, h_min = INT_MAX;

    cudaSafeCall(hipMalloc(&d_max, sizeof(pixel_t)));
    cudaSafeCall(hipMalloc(&d_min, sizeof(pixel_t)));

    cudaSafeCall(hipMemcpy(d_max, &h_max, sizeof(pixel_t), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_min, &h_min, sizeof(pixel_t), hipMemcpyHostToDevice));

    min_max_cuda<<<grid, block, 2 * block.x * block.y * sizeof(pixel_t)>>>(out, nx, ny, d_min, d_max);
    cudaCheckMsg("min_max_cuda launch failed");

    // Copy min/max values back to host
    cudaSafeCall(hipMemcpy(&h_max, d_max, sizeof(pixel_t), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(&h_min, d_min, sizeof(pixel_t), hipMemcpyDeviceToHost));

    // Normalize the result
    normalize_cuda<<<grid, block>>>(out, nx, ny, n, h_min, h_max);
    cudaCheckMsg("normalize_cuda launch failed");

    // Clean up
    cudaSafeCall(hipFree(d_kernel));
    cudaSafeCall(hipFree(d_max));
    cudaSafeCall(hipFree(d_min));
}

__global__ void non_maximum_suppression_kernel(const pixel_t *after_Gx, const pixel_t *after_Gy, const pixel_t *G, pixel_t *nms, const int nx, const int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || y < 1 || x >= nx - 1 || y >= ny - 1)
        return;

    int c = x + y * nx;
    const int nn = c - nx;
    const int ss = c + nx;
    const int ww = c + 1;
    const int ee = c - 1;
    const int nw = nn + 1;
    const int ne = nn - 1;
    const int sw = ss + 1;
    const int se = ss - 1;

    const float dir = (float)(fmod(atan2f(after_Gy[c], after_Gx[c]) + M_PI, M_PI) / M_PI) * 8;

    if (((dir <= 1 || dir > 7) && G[c] > G[ee] && G[c] > G[ww]) || // 0 deg
        ((dir > 1 && dir <= 3) && G[c] > G[nw] && G[c] > G[se]) || // 45 deg
        ((dir > 3 && dir <= 5) && G[c] > G[nn] && G[c] > G[ss]) || // 90 deg
        ((dir > 5 && dir <= 7) && G[c] > G[ne] && G[c] > G[sw]))   // 135 deg
        nms[c] = G[c];
    else
        nms[c] = 0;
}

// edges found in first pass for nms > tmax
void first_edges(const pixel_t *nms, pixel_t *reference,
                 const int nx, const int ny, const int tmax)
{

    size_t c = 1;
    for (int j = 1; j < ny - 1; j++)
    {
        for (int i = 1; i < nx - 1; i++)
        {
            if (nms[c] >= tmax)
            { // trace edges
                reference[c] = MAX_BRIGHTNESS;
            }
            c++;
        }
        c += 2; // because borders are not considered
    }
}


__global__ void first_edges_kernel(const pixel_t *nms, pixel_t *reference,
                  const int nx, const int ny, const int tmax)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels just like in the original CPU version
    if (x < 1 || y < 1 || x >= nx - 1 || y >= ny - 1)
        return;

    size_t c = x + y * nx;
    
    // Same logic as CPU implementation - mark pixels >= tmax as edges
    if (nms[c] >= tmax)
    {
        reference[c] = MAX_BRIGHTNESS;
    }
}

// edges found in after first passes for nms > tmin && neighbor is edge
void hysteresis_edges(const pixel_t *nms, pixel_t *reference,
                      const int nx, const int ny, const int tmin, bool *pchanged)
{
    // Tracing edges with hysteresis . Non-recursive implementation.
    for (int i = 1; i < nx - 1; i++)
    {
        for (int j = 1; j < ny - 1; j++)
        {
            size_t t = i + j * nx;

            int nbs[8];          // neighbours
            nbs[0] = t - nx;     // nn
            nbs[1] = t + nx;     // ss
            nbs[2] = t + 1;      // ww
            nbs[3] = t - 1;      // ee
            nbs[4] = nbs[0] + 1; // nw
            nbs[5] = nbs[0] - 1; // ne
            nbs[6] = nbs[1] + 1; // sw
            nbs[7] = nbs[1] - 1; // se

            if (nms[t] >= tmin && reference[t] == 0)
            {
                for (int k = 0; k < 8; k++)
                    if (reference[nbs[k]] != 0)
                    {
                        reference[t] = MAX_BRIGHTNESS;
                        *pchanged = true;
                    }
            }
        }
    }
}

__global__ void hysteresis_edges_kernel(const pixel_t *nms, pixel_t *reference,
                      const int nx, const int ny, const int tmin, int *changed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels
    if (x < 1 || y < 1 || x >= nx - 1 || y >= ny - 1)
        return;

    size_t t = x + y * nx;

    // Pixels that are above tmin but not yet marked as edges
    if (nms[t] >= tmin && reference[t] == 0)
    {
        // Check all 8 neighboring pixels
        int nbs[8];          // neighbours
        nbs[0] = t - nx;     // nn
        nbs[1] = t + nx;     // ss
        nbs[2] = t + 1;      // ww
        nbs[3] = t - 1;      // ee
        nbs[4] = nbs[0] + 1; // nw
        nbs[5] = nbs[0] - 1; // ne
        nbs[6] = nbs[1] + 1; // sw
        nbs[7] = nbs[1] - 1; // se

        // Check if any neighbor is an edge
        for (int k = 0; k < 8; k++)
        {
            if (reference[nbs[k]] != 0)
            {
                reference[t] = MAX_BRIGHTNESS;
                atomicExch(changed, 1); // Signal that we made a change
                break;
            }
        }
    }
}

__global__ void merge_gradients_kernel(const pixel_t *after_Gx, const pixel_t *after_Gy, pixel_t *G, const int nx, const int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || y < 1 || x >= nx - 1 || y >= ny - 1)
        return;

    int c = x + y * nx;
    G[c] = (pixel_t)(hypot((double)(after_Gx[c]), (double)(after_Gy[c])));
}

// canny edge detector code to run on the GPU
void cannyDevice(const int *h_idata, const int w, const int h,
                 const int tmin, const int tmax,
                 const float sigma,
                 int *h_odata)
{
    const int nx = w;
    const int ny = h;
    const size_t image_size = nx * ny * sizeof(pixel_t);
    const int conv_kernel_size = 3;

    // Device memory pointers - only allocate what we actually need
    pixel_t *d_input = NULL, *d_temp = NULL, *d_Gx = NULL, *d_Gy = NULL;
    pixel_t *d_nms = NULL, *d_G = NULL, *d_reference = NULL;
    float *d_kernel = NULL;
    int *d_changed = NULL;

    // Allocate device memory
    cudaSafeCall(hipMalloc(&d_input, image_size));
    cudaSafeCall(hipMalloc(&d_temp, image_size));  // Temporary buffer for Gaussian output
    cudaSafeCall(hipMalloc(&d_kernel, sizeof(float) * conv_kernel_size * conv_kernel_size));
    cudaSafeCall(hipMalloc(&d_G, image_size));
    cudaSafeCall(hipMalloc(&d_Gx, image_size));
    cudaSafeCall(hipMalloc(&d_Gy, image_size));
    cudaSafeCall(hipMalloc(&d_nms, image_size));
    cudaSafeCall(hipMalloc(&d_reference, image_size));
    cudaSafeCall(hipMalloc(&d_changed, sizeof(int)));

    // Copy input data to device
    cudaSafeCall(hipMemcpy(d_input, h_idata, image_size, hipMemcpyHostToDevice));

    // Initialize reference buffer to zeros
    cudaSafeCall(hipMemset(d_reference, 0, image_size));

    // Apply Gaussian filter (now fully on GPU)
    gaussian_filter_cuda(d_input, d_temp, nx, ny, sigma);

    // Set up grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    // Compute gradients using Sobel operators
    const float Gx[] = {-1, 0, 1,
                        -2, 0, 2,
                        -1, 0, 1};
    cudaSafeCall(hipMemcpy(d_kernel, Gx, conv_kernel_size * conv_kernel_size * sizeof(float), hipMemcpyHostToDevice));

    // X gradient
    convolution_cuda_kernel<<<gridDim, blockDim>>>(d_temp, d_Gx, d_kernel, nx, ny, conv_kernel_size);
    cudaCheckMsg("convolution_cuda_kernel X launch failed");

    const float Gy[] = {1, 2, 1,
                        0, 0, 0,
                        -1, -2, -1};
    cudaSafeCall(hipMemcpy(d_kernel, Gy, conv_kernel_size * conv_kernel_size * sizeof(float), hipMemcpyHostToDevice));

    // Y gradient
    convolution_cuda_kernel<<<gridDim, blockDim>>>(d_temp, d_Gy, d_kernel, nx, ny, conv_kernel_size);
    cudaCheckMsg("convolution_cuda_kernel Y launch failed");

    // Merge gradients to compute magnitude
    merge_gradients_kernel<<<gridDim, blockDim>>>(d_Gx, d_Gy, d_G, nx, ny);
    cudaCheckMsg("merge_gradients_kernel launch failed");

    // Non-maximum suppression
    non_maximum_suppression_kernel<<<gridDim, blockDim>>>(d_Gx, d_Gy, d_G, d_nms, nx, ny);
    cudaCheckMsg("non_maximum_suppression_kernel launch failed");

    // First edge detection (pixels >= tmax)
    first_edges_kernel<<<gridDim, blockDim>>>(d_nms, d_reference, nx, ny, tmax);
    cudaCheckMsg("first_edges_kernel launch failed");

    // Hysteresis edge linking
    int h_changed;
    do {
        // Reset changed flag
        h_changed = 0;
        cudaSafeCall(hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice));
        
        // Run hysteresis kernel
        hysteresis_edges_kernel<<<gridDim, blockDim>>>(d_nms, d_reference, nx, ny, tmin, d_changed);
        cudaCheckMsg("hysteresis_edges_kernel launch failed");
        
        // Check if any changes were made
        cudaSafeCall(hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost));
    } while (h_changed);

    // Copy result back to host
    cudaSafeCall(hipMemcpy(h_odata, d_reference, image_size, hipMemcpyDeviceToHost));

    // Free all device memory
    cudaSafeCall(hipFree(d_input));
    cudaSafeCall(hipFree(d_temp));
    cudaSafeCall(hipFree(d_kernel));
    cudaSafeCall(hipFree(d_G));
    cudaSafeCall(hipFree(d_Gx));
    cudaSafeCall(hipFree(d_Gy));
    cudaSafeCall(hipFree(d_nms));
    cudaSafeCall(hipFree(d_reference));
    cudaSafeCall(hipFree(d_changed));
}