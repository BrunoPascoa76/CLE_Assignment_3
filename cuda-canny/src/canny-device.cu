#include "hip/hip_runtime.h"

// CLE 24'25
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// utilities for safe cuda api calls copied from cuda sdk.

#define cudaSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
#define cudaCheckMsg(msg)           __cudaGetLastError  (msg, __FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
      if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error %d: %s.\n",
                  file, line, (int)err, hipGetErrorString(err));
          exit(-1);
      }
}

inline void __cudaGetLastError( const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : cutilCheckMsg() CUTIL CUDA error : %s : (%d) %s.\n",
                file, line, errorMessage, (int)err, hipGetErrorString( err ));
        exit(-1);
    }
}

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255

// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

// include image functions
#include "image.c"

// convolution of in image to out image using kernel of kn width
void convolution(const pixel_t *in, pixel_t *out, const float *kernel,
                 const int nx, const int ny, const int kn)
{
    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);
    const int khalf = kn / 2;

    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {
            float pixel = 0.0;
            size_t c = 0;
            for (int j = -khalf; j <= khalf; j++)
                for (int i = -khalf; i <= khalf; i++) {
                    pixel += in[(n - j) * nx + m - i] * kernel[c];
                    c++;
                }

            out[n * nx + m] = (pixel_t)pixel;
        }
}

__global__ void convolution_cuda_kernel(const pixel_t *in, pixel_t *out, const float *kernel, const int nx, const int ny, const int kn) {
    // get current coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int khalf = kn / 2;

    if (x >=nx || y>=ny)
        return; // out of bounds (if image size is not multiple of 16)

    float sum= 0.0f;

    //now do the actual convolution for this pixel
    for(int ky=-khalf; ky<=khalf; ky++){
        for(int kx=-khalf; kx<=khalf; kx++){
            int ix= x+kx;
            int iy= y+ky;

            if(ix>=0 && ix<nx && iy>=0 && iy<ny){ //if the current kernel point is whithin bounds...
                float val= in[iy*nx+ix];
                float weight= kernel[(ky+khalf)*kn+ (kx+khalf)];
                sum+= val*weight;
            }
        }
    }

    out[y*nx+x]= (pixel_t)sum;
}

// determines min and max of in image
void min_max(const pixel_t *in, const int nx, const int ny, pixel_t *pmin, pixel_t *pmax)
{
    int min = INT_MAX, max = -INT_MAX;

    for (int m = 0; m < nx; m++)
        for (int n = 0; n < ny ; n++) {
            int pixel = in[n*nx + m];
            if (pixel < min)
                min = pixel;
            if (pixel > max)
                max = pixel;
        }
    *pmin = min; *pmax = max;
}

// normalizes inout image using min and max values
void normalize(  pixel_t *inout,
               const int nx, const int ny, const int kn,
               const int min, const int max)
{
    const int khalf = kn / 2;

    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {

            pixel_t pixel = MAX_BRIGHTNESS * ((int)inout[n * nx + m] -(float) min) / ((float)max - (float)min);
            inout[n * nx + m] = pixel;
        }
}


/*
 * gaussianFilter:
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 * determine size of kernel (odd #)
 * 0.0 <= sigma < 0.5 : 3
 * 0.5 <= sigma < 1.0 : 5
 * 1.0 <= sigma < 1.5 : 7
 * 1.5 <= sigma < 2.0 : 9
 * 2.0 <= sigma < 2.5 : 11
 * 2.5 <= sigma < 3.0 : 13 ...
 * kernelSize = 2 * int(2*sigma) + 3;
 */
void gaussian_filter(const pixel_t *in, pixel_t *out,
                     const int nx, const int ny, const float sigma)
{
    const int n = 2 * (int)(2 * sigma) + 3;
    const float mean = (float)floor(n / 2.0);
    float kernel[n * n]; // variable length array

    fprintf(stderr, "gaussian_filter: kernel size %d, sigma=%g\n",
            n, sigma);
    size_t c = 0;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            kernel[c] = exp(-0.5 * (pow((i - mean) / sigma, 2.0) +
                            pow((j - mean) / sigma, 2.0)))
                / (2 * M_PI * sigma * sigma);
            c++;
        }

    convolution(in, out, kernel, nx, ny, n);
    pixel_t max, min;
    min_max(out, nx, ny, &min, &max);
    normalize(out, nx, ny, n, min, max);
}

// Canny non-maximum suppression
void non_maximum_supression(const pixel_t *after_Gx, const pixel_t * after_Gy, const pixel_t *G, pixel_t *nms,
                            const int nx, const int ny)
{
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            const int nn = c - nx;
            const int ss = c + nx;
            const int ww = c + 1;
            const int ee = c - 1;
            const int nw = nn + 1;
            const int ne = nn - 1;
            const int sw = ss + 1;
            const int se = ss - 1;

            const float dir = (float)(fmod(atan2(after_Gy[c],
                                                 after_Gx[c]) + M_PI,
                                           M_PI) / M_PI) * 8;

            if (((dir <= 1 || dir > 7) && G[c] > G[ee] &&
                G[c] > G[ww]) || // 0 deg
                ((dir > 1 && dir <= 3) && G[c] > G[nw] &&
                G[c] > G[se]) || // 45 deg
                ((dir > 3 && dir <= 5) && G[c] > G[nn] &&
                G[c] > G[ss]) || // 90 deg
                ((dir > 5 && dir <= 7) && G[c] > G[ne] &&
                G[c] > G[sw]))   // 135 deg
                nms[c] = G[c];
            else
                nms[c] = 0;
        }
}

// edges found in first pass for nms > tmax
void first_edges(const pixel_t *nms, pixel_t *reference,
                 const int nx, const int ny, const int tmax)
{

    size_t c = 1;
    for (int j = 1; j < ny - 1; j++) {
        for (int i = 1; i < nx - 1; i++) {
            if (nms[c] >= tmax) { // trace edges
                reference[c] = MAX_BRIGHTNESS;
            }
            c++;
        }
        c+=2; // because borders are not considered
    }
}


// edges found in after first passes for nms > tmin && neighbor is edge
void hysteresis_edges(const pixel_t *nms, pixel_t *reference,
                      const int nx, const int ny, const int tmin, bool *pchanged)
{
    // Tracing edges with hysteresis . Non-recursive implementation.
    for (int i = 1; i < nx - 1; i++) {
        for (int j = 1; j < ny - 1; j++) {
            size_t t = i + j * nx;

            int nbs[8]; // neighbours
            nbs[0] = t - nx;     // nn
            nbs[1] = t + nx;     // ss
            nbs[2] = t + 1;      // ww
            nbs[3] = t - 1;      // ee
            nbs[4] = nbs[0] + 1; // nw
            nbs[5] = nbs[0] - 1; // ne
            nbs[6] = nbs[1] + 1; // sw
            nbs[7] = nbs[1] - 1; // se

            if (nms[t] >= tmin && reference[t] == 0) {
                for(int k = 0; k < 8; k++)
                    if (reference[nbs[k]] != 0) {
                        reference[t] = MAX_BRIGHTNESS;
                        *pchanged = true;
                    }
            }
        }
    }
}


// canny edge detector code to run on the GPU
void cannyDevice( const int *h_idata, const int w, const int h,
                 const int tmin, const int tmax,
                 const float sigma,
                 int * h_odata)
{
    const int nx = w;
    const int ny = h;
    const int conv_kernel_size = 3;

    pixel_t *G        = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gx = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gy = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *nms      = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));

    pixel_t *input=NULL, *output=NULL, *d_Gx=NULL, *d_Gy=NULL, *d_nms=NULL, *d_G=NULL;
    float *kernel=NULL;

    cudaSafeCall(hipMalloc(&input, sizeof(pixel_t) * nx * ny));
    cudaSafeCall(hipMalloc(&output, sizeof(pixel_t) * nx * ny));
    cudaSafeCall(hipMalloc(&kernel, sizeof(float) * conv_kernel_size * conv_kernel_size));
    cudaSafeCall(hipMalloc(&d_G, sizeof(pixel_t) * nx * ny));
    cudaSafeCall(hipMalloc(&d_Gx, sizeof(pixel_t) * nx * ny));
    cudaSafeCall(hipMalloc(&d_Gy, sizeof(pixel_t) * nx * ny));
    cudaSafeCall(hipMalloc(&d_nms, sizeof(pixel_t) * nx * ny));
    

    if (G == NULL || after_Gx == NULL || after_Gy == NULL ||
        nms == NULL || h_odata == NULL) {
        fprintf(stderr, "canny_edge_detection:"
                " Failed memory allocation(s).\n");
        exit(1);
    }

    // Gaussian filter
    gaussian_filter(h_idata, h_odata, nx, ny, sigma);

    cudaSafeCall(hipMemcpy(input,  h_odata, nx * ny * sizeof(pixel_t), hipMemcpyHostToDevice));

    dim3 blockDim(16,16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    // x gradient convolution

    const float Gx[] = {-1, 0, 1,
        -2, 0, 2,
        -1, 0, 1};
    cudaSafeCall(hipMemcpy(kernel, Gx,      conv_kernel_size * conv_kernel_size * sizeof(float), hipMemcpyHostToDevice));
    
    //call for x direction
    convolution_cuda_kernel<<<gridDim, blockDim>>>(input, output, kernel, nx, ny, conv_kernel_size);

    cudaCheckMsg("convolution_cuda_kernel X launch failed");
    cudaSafeCall(hipDeviceSynchronize());

    //copy over to temporary buffer
    cudaSafeCall(hipMemcpy(after_Gx, output, nx*ny * sizeof(pixel_t), hipMemcpyDeviceToHost));

    
    const float Gy[] = { 1, 2, 1,
        0, 0, 0,
        -1,-2,-1};
    cudaSafeCall(hipMemcpy(kernel, Gy, conv_kernel_size * conv_kernel_size * sizeof(float), hipMemcpyHostToDevice));

    convolution_cuda_kernel<<<gridDim, blockDim>>>(input, output, kernel, nx, ny, conv_kernel_size);
    cudaCheckMsg("convolution_cuda_kernel Y launch failed");
    cudaSafeCall(hipDeviceSynchronize());

    //copy over results
    cudaSafeCall(hipMemcpy(after_Gy, output, nx*ny * sizeof(pixel_t), hipMemcpyDeviceToHost));

    // Merging gradients
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            G[c] = (pixel_t)(hypot((double)(after_Gx[c]), (double)( after_Gy[c]) ));
        }

    // Non-maximum suppression, straightforward implementation.
    non_maximum_supression(after_Gx, after_Gy, G, nms, nx, ny);

    // edges with nms >= tmax
    memset(h_odata, 0, sizeof(pixel_t) * nx * ny);
    first_edges(nms, h_odata, nx, ny, tmax);

    // edges with nms >= tmin && neighbor is edge
    bool changed;
    do {
        changed = false;
        hysteresis_edges(nms, h_odata, nx, ny, tmin, &changed);
    } while (changed==true);

    // Free device memory
    cudaSafeCall(hipFree(input));
    cudaSafeCall(hipFree(output));
    cudaSafeCall(hipFree(kernel));
    cudaSafeCall(hipFree(d_G));
    cudaSafeCall(hipFree(d_Gx));
    cudaSafeCall(hipFree(d_Gy));
    cudaSafeCall(hipFree(d_nms));

    free(after_Gx);
    free(after_Gy);
    free(G);
    free(nms);
}
